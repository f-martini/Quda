
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int N = 5;
    float A[] = {1, 2, 3, 4, 5};
    float B[] = {1, 2, 3, 4, 5};
    float C[] = {0, 0, 0, 0, 0};

    size_t size = N * sizeof(float);

    float *h_A = A;
    float *h_B = B;
    float *h_C = C;

    // Allocate vectors in device memory
    float *d_A;
    hipMalloc(&d_A, size);
    float *d_B;
    hipMalloc(&d_B, size);
    float *d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = N;
    int blocksPerGrid =
        (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("%.2f ", h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}